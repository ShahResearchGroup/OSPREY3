
#include <hip/hip_runtime.h>

// compile with
// nvcc -fatbin -arch=compute_20 "kernelSource/test.cu" -o "kernelBinaries/test.bin"

extern "C" __global__ void add(int n, double *a, double *b, double *out) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (i < n) {
		out[i] = a[i] + b[i];
	}
}
