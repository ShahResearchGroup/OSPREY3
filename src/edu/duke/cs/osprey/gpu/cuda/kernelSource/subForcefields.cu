
// compile with:
// nvcc -fatbin -arch=compute_20 "kernelSource/subForcefields.cu" -o "kernelBinaries/subForcefields.bin"


#include <hip/hip_runtime.h>
#include <stdio.h>


// NOTE: should be identical to forcefield.cu
typedef struct __align__(8) {
	int numPairs; // @ 0
	int num14Pairs; // @ 4
	double coulombFactor; // @ 8
	double scaledCoulombFactor; // @ 16
	double solvCutoff2; // @ 24
	bool useDistDepDielec; // @ 32
	bool useHEs; // @ 33
	bool useHVdw; // @ 34
	bool doEnergy; // @ 35
} ForcefieldArgs;
// sizeof = 36


// jesus, CUDA... no libraries for vector math? what gives??

__device__ void set(double2 &v, double x, double y) {
	v.x = x;
	v.y = y;
}

__device__ void set(double3 &v, double x, double y, double z) {
	v.x = x;
	v.y = y;
	v.z = z;
}

__device__ void sub(double3 &a, double3 &b) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
}

__device__ void add(double3 &a, double3 &b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

__device__ double dot(double2 &a, double2 &b) {
	return a.x*b.x + a.y*b.y;
}

__device__ double dot(double3 &a, double3 &b) {
	return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__ void cross(double3 &out, double3 &a, double3 &b) {
	out.x = a.y*b.z - a.z*b.y;
	out.y = a.z*b.x - a.x*b.z;
	out.z = a.x*b.y - a.y*b.x;
}

__device__ double lengthSq(double2 &v) {
	return dot(v, v);
}

__device__ double lengthSq(double3 &v) {
	return dot(v, v);
}

__device__ double length(double2 &v) {
	return sqrt(lengthSq(v));
}

__device__ double length(double3 &v) {
	return sqrt(lengthSq(v));
}

__device__ void negate(double3 &v) {
	v.x = -v.x;
	v.y = -v.y;
	v.z = -v.z;
}

__device__ void mult(double3 &v, double c) {
	v.x *= c;
	v.y *= c;
	v.z *= c;
}

__device__ void div(double3 &v, double c) {
	v.x /= c;
	v.y /= c;
	v.z /= c;
}

__device__ void normalize(double2 &v) {
	double l = length(v);
	v.x /= l;
	v.y /= l;
}

__device__ void normalize(double3 &v) {
	double l = length(v);
	v.x /= l;
	v.y /= l;
	v.z /= l;
}

__device__ void rotateVec(double3 &v, double3 &x, double3 &y, double3 &z) {
	set(v,
		v.x*x.x + v.y*y.x + v.z*z.x,
		v.x*x.y + v.y*y.y + v.z*z.y,
		v.x*x.z + v.y*y.z + v.z*z.z
	);
}

__device__ void rotateVecInverse(double3 &v, double3 &x, double3 &y, double3 &z) {
	set(v,
		dot(v, x),
		dot(v, y),
		dot(v, z)
	);
}

__device__ void rotateVecZ(double3 &v, double &sinTheta, double &cosTheta) {
	double vx = v.x*cosTheta - v.y*sinTheta;
	double vy = v.x*sinTheta + v.y*cosTheta;
	v.x = vx;
	v.y = vy;
}

__device__ double3 readCoord(double *coords, int i) {
	int i3 = i*3;
	return make_double3(coords[i3], coords[i3 + 1], coords[i3 + 2]);
}

__device__ void writeCoord(double *coords, int i, double3 &val) {
	int i3 = i*3;
	coords[i3] = val.x;
	coords[i3 + 1] = val.y;
	coords[i3 + 2] = val.z;
}


__device__ int divUp(int a, int b) {
	// ie.,  ceil(a/b)
	return (a + b - 1)/b;
}

__device__ int getAtomIndex(int flags) {
	return abs(flags) - 1;
}

__device__ bool isHydrogen(int flags) {
	return flags > 0;
}

__device__ double calcPairEnergy(
	const double *coords,
	const int *atomFlags,
	const double *precomputed,
	const ForcefieldArgs *args,
	const int i, // pairIndex
	const bool is14Pair
) {

	// start with zero energy
	double energy = 0;
	
	// read atom flags and calculate all the things that use the atom flags in this scope
	bool bothHeavy;
	double r2 = 0;
	{
		int atom1Flags, atom2Flags;
		{
			int i2 = i*2;
			atom1Flags = atomFlags[i2];
			atom2Flags = atomFlags[i2 + 1];
		}
		
		bothHeavy = !isHydrogen(atom1Flags) && !isHydrogen(atom2Flags);
		
		// calculate the squared radius
		int atom1Index3 = getAtomIndex(atom1Flags)*3;
		int atom2Index3 = getAtomIndex(atom2Flags)*3;
		double d;
		d = coords[atom1Index3] - coords[atom2Index3];
		r2 += d*d;
		d = coords[atom1Index3 + 1] - coords[atom2Index3 + 1];
		r2 += d*d;
		d = coords[atom1Index3 + 2] - coords[atom2Index3 + 2];
		r2 += d*d;
	}
	
	int i9 = i*9;
	
	// calculate electrostatics
	if (bothHeavy || args->useHEs) {
	
		double esEnergy = is14Pair ? args->scaledCoulombFactor : args->coulombFactor;
		
		{
			double charge = precomputed[i9 + 2];
			esEnergy *= charge;
		}
		
		{
			esEnergy /= args->useDistDepDielec ? r2 : sqrt(r2);
		}
		
		energy += esEnergy;
	}
	
	// calculate vdw
	if (bothHeavy || args->useHVdw) {
		
		double Aij, Bij;
		{
			Aij = precomputed[i9];
			Bij = precomputed[i9 + 1];
		}
		
		// compute vdw
		double r6 = r2*r2*r2;
		double r12 = r6*r6;
		energy += Aij/r12 - Bij/r6;
	}
	
	// calculate solvation
	if (bothHeavy && r2 < args->solvCutoff2) {
			
		double r = sqrt(r2);
		{
			double lambda1 = precomputed[i9 + 3];
			double radius1 = precomputed[i9 + 4];
			double alpha1 = precomputed[i9 + 5];
			double Xij = (r - radius1)/lambda1;
			energy -= alpha1*exp(-Xij*Xij)/r2;
		}
		{
			double lambda2 = precomputed[i9 + 6];
			double radius2 = precomputed[i9 + 7];
			double alpha2 = precomputed[i9 + 8];
			double Xji = (r - radius2)/lambda2;
			energy -= alpha2*exp(-Xji*Xji)/r2;
		}
	}
	
	return energy;
}

__device__ void blockSum(double *scratch, double *out) {

	int threadId = threadIdx.x;
	int numThreads = blockDim.x;
	
	// compute the energy sum in SIMD-style
	// see url for a tutorial on GPU reductions:
	// http://developer.amd.com/resources/articles-whitepapers/opencl-optimization-case-study-simple-reductions/

	__syncthreads();
	
	for (int offset = 1; offset < numThreads; offset <<= 1) {
	
		// sum this level of the reduction tree
		int mask = (offset << 1) - 1;
		if ((threadId & mask) == 0) {
			scratch[threadId] += scratch[threadId + offset];
		}
		
		__syncthreads();
	}
	
	// finally, if we're the 0 thread, write the summed energy for this work group
	if (threadId == 0) {
		out[blockIdx.x] = scratch[0];
	}
}

__device__ void pose(double *coords, const int *dihedralIndices, const int numRotatedIndices, const int *rotatedIndices, const double dihedralRadians) {

	if (threadIdx.x >= numRotatedIndices) {
		return;
	}
	
	// get the four atom positions: a, b, c, d
	double3 a = readCoord(coords, dihedralIndices[0]);
	double3 b = readCoord(coords, dihedralIndices[1]);
	double3 c = readCoord(coords, dihedralIndices[2]);
	double3 d = readCoord(coords, dihedralIndices[3]);
	
	// translate so everything is centered on b
	sub(a, b);
	sub(c, b);
	sub(d, b);
	
	// build a right orthonormal matrix [rx,ry,rz] where z is bc and ba points along x
	double3 rz = c;
	normalize(rz);
	
	double3 rx = c;
	mult(rx, dot(a, c)/dot(c, c));
	negate(rx);
	add(rx, a);
	normalize(rx);
	
	double3 ry;
	cross(ry, rz, rx);
	
	// use r^{-1} to rotate d into our axis-aligned space
	rotateVecInverse(d, rx, ry, rz);
	
	// look at the x,y coords of d to get the dihedral angle
	double2 cossin = make_double2(d.x, d.y);
	normalize(cossin);
	double currentSin = cossin.y;
	double currentCos = cossin.x;
	
	// get the delta dihedral
	double newSin, newCos;
	sincos(dihedralRadians, &newSin, &newCos);
	double deltaSin = newSin*currentCos - newCos*currentSin;
	double deltaCos = newCos*currentCos + newSin*currentSin;
	
	// rotate atoms in parallel
	int index = rotatedIndices[threadIdx.x];
	double3 p = readCoord(coords, index);
	sub(p, b);
	rotateVecInverse(p, rx, ry, rz);
	rotateVecZ(p, deltaSin, deltaCos);
	rotateVec(p, rx, ry, rz);
	add(p, b);
	writeCoord(coords, index, p);
}

extern "C" __global__ void calcEnergies(
	const double *coords,
	const int numCoords,
	const int *atomFlags,
	const double *precomputed,
	const ForcefieldArgs *args,
	const double *dihedrals,
	const int *dihedralIndices,
	const int *rotatedIndices,
	const int *subsets,
	const int *subsetOffsets,
	double *out
) {

	int threadId = threadIdx.x;
	int numThreads = blockDim.x;
	int blockId = blockIdx.x;
	
	// partition shared memory
	extern __shared__ double shared[]; // NOTE: can't declare as pointer, must be array
	double *threadEnergies = shared;
	double *sharedCoords = shared + numThreads;
	
	// read subset info
	// NOTE: read num pairs from subset info, not forcefield args
	int subsetOffset = subsetOffsets[blockId];
	int numPairs = subsets[subsetOffset];
	int num14Pairs = subsets[subsetOffset + 1];
	
	// copy coords to shared mem
	// TODO: can't fit the whole molecule in shared mem in general!
	// need to just copy the parts we change
	{
		int threadCoords = divUp(numCoords, numThreads);
		int i = threadId*threadCoords;
		int stopI = min(numCoords, (threadId + 1)*threadCoords);
		for (; i < stopI; i++) {
			sharedCoords[i] = coords[i];
		}
	}
	__syncthreads();
	
	// pose the protein
	int dof = blockId/3;
	const int *dihedralIndicesDof = dihedralIndices + dof*4;
	int maxNumRotatedAtoms = rotatedIndices[0];
	const int *rotatedIndicesDof = rotatedIndices + 1 + dof*(maxNumRotatedAtoms + 1);
	int numRotatedAtoms = rotatedIndicesDof[0];
	rotatedIndicesDof++;
	double dihedralRadians = dihedrals[blockId];
	pose(sharedCoords, dihedralIndicesDof, numRotatedAtoms, rotatedIndicesDof, dihedralRadians);
	__syncthreads();
	
	// partition atom pairs among threads
	int threadPairs = divUp(numPairs, numThreads);
	int i = threadId*threadPairs;
	int stopI = min(numPairs, (threadId + 1)*threadPairs);
	
	// calc energies for atom pairs
	double energy = 0;
	for (; i < stopI; i++) {
		int pairIndex = subsets[subsetOffset + 2 + i];
		bool is14Pair = i > num14Pairs;
		energy += calcPairEnergy(sharedCoords, atomFlags, precomputed, args, pairIndex, is14Pair);
	}
	threadEnergies[threadId] = energy;
	
	// sum energies from all threads
	blockSum(threadEnergies, out);
}
